#include "hip/hip_runtime.h"
#include <iostream>

/*
kernel              - function that runs on GPU
__global__ keyword  - it tells CUDA C++ compiler that the function
runs in GPU and can be called from CPU code
*/
__global__ void vectorAdd(int N, float* x, float* y){
    for(int i = 0; i < N; i++){
        y[i] = x[i] + y[i];
    }
}

int main(){
    int N = 1<<20;

    //float *x = new float[N];
    //float *y = new float[N];
    /*
    We need to keep the data in memory which is accessible by GPU.
    CUDA provides "Unified Memory" space which is accessible by both CPU and GPU
    hipMallocManaged       - allocate memory in Unified Memory space
    */
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y vectors on host
    for(int i = 0; i < N; i++){
        x[i] = float(i);
        y[i] = float(i);
    }

    //vectorAdd(N, x, y);
    /* 
    launcing kernel on GPU are done using <<<>>>
    */
    vectorAdd <<<1, 1>>> (N, x, y);

    /*
    cudaDeviceSyncronize        - block CPU execution until kernel execution is done
    */
    cudaDeviceSyncronize();
    
    // verify the operation is successful
    float maxError = 0.0f;
    for(int i = 0; i < N; i++){
        float error = (y[i] - 2*x[i]) > 0 ? y[i] - 2*x[i] : 2*x[i] - y[i];
        maxError = maxError > error ? maxError : error;
    }
    std::cout << "Maximum error " << maxError << std::endl;

    //delete [] x;
    //delete [] y;
    /* 
    hipFree        - free memory from Unified Memory space
    */
    hipFree(x);
    hipFree(y);

    return 0;
}